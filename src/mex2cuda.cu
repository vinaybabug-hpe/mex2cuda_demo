#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : ensembleClust_pall_cuda.cu
 Author      : Vinay B Gavirangaswamy
 Version     :
 Copyright   :  This program is free software: you can redistribute it and/or modify
    			it under the terms of the GNU General Public License as published by
    			the Free Software Foundation, either version 3 of the License, or
    			(at your option) any later version.

    			This program is distributed in the hope that it will be useful,
    			but WITHOUT ANY WARRANTY; without even the implied warranty of
    			MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    			GNU General Public License for more details.


    			You should have received a copy of the GNU General Public License
    			along with this program.  If not, see <http://www.gnu.org/licenses/>.
 Description : CUDA related code goes into this file. You can customize it according to project needs. 
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "common/wrapperFuncs.h"

__global__ void cudaKernel();

/*
 * Wrapper function
 */

void mex2CudaWrapper(){

	printf("Hello World from cuda kernel\n");

	cudaKernel<<<1, 1>>>();

}

/*
 * Host code
 */
__global__ void cudaKernel(){

	int tid = blockIdx.x;



}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
/*static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}*/

